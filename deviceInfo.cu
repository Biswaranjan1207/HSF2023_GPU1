#include <stdio.h>

int main()
{
 int nDevices;
 hipGetDeviceCount(&nDevices);
 printf("--------------------------------------------------------------------------------------------------------------------------------\n");
 for (int i = 0; i < nDevices; i++)
 {
   hipDeviceProp_t prop;
   hipGetDeviceProperties(&prop, i);
   printf("Device Number: %d\n", i);
   printf("  Device name: %s\n", prop.name);
   printf("  Memory Clock Rate (KHz): %d\n",prop.memoryClockRate);
   printf("  Memory Bus Width (bits): %d\n",prop.memoryBusWidth);
   printf("  Compute capability: %d.%d\n",prop.major,prop.minor);
   printf("  Peak Memory Bandwidth (GB/s): %f\n\n",2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
   printf("  Number of SMs: %d\n", prop.multiProcessorCount);
   printf("  Maximum grid dimensions:  %d,%d,%d\n",prop.maxGridSize[0] ,prop.maxGridSize[1] ,prop.maxGridSize[2]);
   printf("  Warp size  %d\n",prop.warpSize);
   printf("  Max # of threads / block: %d\n",prop.maxThreadsPerBlock);
   printf("  Max size of a block blockDim.x : %d, .y : %d, .z : %d \n",prop.maxThreadsDim[0], prop.maxThreadsDim[1],prop.maxThreadsDim[2]);
 }
 printf("--------------------------------------------------------------------------------------------------------------------------------\n");
}

